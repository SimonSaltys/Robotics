#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(Mat* left, Mat* right, 
	Mat* depthImage, double maxDistance, int rows, int cols){

const int windowWidth = 11;
const int halfWindow = (windowWidth - 1) / 2
double maxDisparity = 71;
double focalLength = -1; // pixels
double baseLine = 60.0 // mm
double disparity;
double distance;
unsigned char leftPixel;
unsigned char rightPixel;

// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

int sumSqDiff;
int minSumSqDiff = INT_MAX;

	//Safety check if we are trying to access something not in the image
	if(row < halfWindow || row > rows-halfWindow ||
	   col < halfWindow || col > cols-halfWindow)
		return;

	for(int k = 0; k < maxDisparity; k++) {
		sumSqDiff = 0;

		for(int i = -halfWindow; i < halfWindow+1;i++)
			for(int j = -halfWindow; i < halfWindow+1; j++) {
				leftPixle = left[(row+i) * cols + (col+j)];
				rightPixle = right[(row+i)* cols+ (col-k+j)];

				int diff = leftPixle - rightPixle;
				sumSqDiff += diff*diff
			}

		if(sumSqDiff < minSumSqDiff) {
			minSumSqDiff = sumSqDiff;
			disparity = k;
		}
	}

	if(disparity > 0) {
		distance = baseLine*focalLength/disparity;
        int index = rows*cols+col;
        
		depthImage[index] = (unsigned char) (255*distance/maxDistance);
		if(depthImage[index] > 255)
			depthImage[index] = (unsigned char) 255;
		else
			//the thing is super far away make it black
			depthImage[index] = 0;
	}
}